/* -------------------------------
 *     Main of the Quakins-X Code
 * ------------------------------- */

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/gather.h>
#include <cstdio>
#include <mpi.h>
#include <nccl.h>
#include <hipfft/hipfftXt.h>

#include "include/initialization.hpp"
#include "include/PhaseSpaceInitialization.hpp"
#include "include/Timer.hpp"
#include "include/ReorderCopy.hpp"
#include "include/FreeStream.hpp"
#include "include/BoundaryCondition.hpp"
#include "include/Integrator.hpp"
#include "include/PoissonSolver.hpp"
#include "include/util.hpp"


using Nums = std::size_t;
using Real = float;


constexpr Nums dim = 4;

#define MCW MPI_COMM_WORLD

int main(int argc, char* argv[]) {

  int mpi_rank, mpi_size, local_rank=0;
  MPI_Init(&argc, &argv);
  MPI_Comm_rank(MCW, &mpi_rank);
  MPI_Comm_size(MCW, &mpi_size);

  uint64_t host_hashs[mpi_size];
  char hostname[1024];
  getHostName(hostname, 1024);
  host_hashs[mpi_rank] = getHostHash(hostname);
  MPI_Allgather(MPI_IN_PLACE,0,MPI_DATATYPE_NULL,host_hashs,
                sizeof(uint64_t), MPI_BYTE, MCW);
  for (int p=0; p<mpi_size; p++) {
    if (p==mpi_rank) break;
    if (host_hashs[p]==host_hashs[mpi_rank]) local_rank++;
  }

  ncclUniqueId nccl_id;
  ncclComm_t comm;
  hipStream_t s;

  if (mpi_rank==0) ncclGetUniqueId(&nccl_id);
  MPI_Bcast((void*)&nccl_id,sizeof(nccl_id),MPI_BYTE,0,MCW);
  
  hipSetDevice(local_rank);
  hipStreamCreate(&s);
  
  ncclCommInitRank(&comm,mpi_size,nccl_id,mpi_rank);

  Parameters<Nums,Real,dim> *p = 
             new Parameters<Nums,Real,dim>;

  try { quakins::init(p, mpi_rank); }
  catch (std::invalid_argument& e) 
  {
    std::cerr << e.what() << std::endl;
#ifndef ANYHOW
    return -1;
#endif
  }
  
  Nums nx1=p->n[2], nx2=p->n[3];
  Nums nx1bd=p->n_ghost[2], nx2bd=p->n_ghost[3];
  Nums nx1tot=p->n_tot[2], nx2tot=p->n_tot_local[3];
  Nums nv1=p->n[0], nv2=p->n[1];
  Nums nxtot = nx1tot*nx2tot;
  Nums comm_size = p->n_ghost[3]*nx1tot*nv1*nv2;
  Nums dens_size = nx1tot*nx2/p->n_dev;
  
  Real v1min=p->low_bound[0], v1max=p->up_bound[0];
  Real v2min=p->low_bound[1], v2max=p->up_bound[1];
  Real x1min=p->low_bound[2], x1max=p->up_bound[2];
  Real x2min=p->low_bound[3], x2max=p->up_bound[3];

  thrust::device_vector<Real> 
    l_send_buff(comm_size), l_recv_buff(comm_size), 
    r_send_buff(comm_size), r_recv_buff(comm_size); 
  thrust::device_vector<Real> 
    f_e(p->n_1d_per_dev), f_e_buff(p->n_1d_per_dev);
  thrust::device_vector<Real> 
    intg_buff(nxtot*nv2), dens_e(nxtot);
  thrust::device_vector<Real> 
    dens_e_tot(nx1tot*nx2), dens_e_tot_buff(nx1tot*nx2), pote_tot(nx1*nx2);
  thrust::host_vector<Real> _dens_e_tot(nx1*nx2), _pote_tot(nx1*nx2);

  std::array<Nums,4> order1 = {2,3,1,0},
                     order2 = {1,0,3,2},
                     order3 = {2,3,1,0};

  std::array<Nums,4> n_now_1 = p->n_tot_local;
  std::array<Nums,4> n_now_2, n_now_3, n_now_4;
  quakins::ReorderCopy<Nums,Real,dim> copy1(n_now_1,order1);
  thrust::gather(order1.begin(),order1.end(),
                 n_now_1.begin(), n_now_2.begin());
  quakins::ReorderCopy<Nums,Real,dim> copy2(n_now_2,order2);
  thrust::gather(order2.begin(),order2.end(),
                 n_now_2.begin(), n_now_3.begin());
  quakins::ReorderCopy<Nums,Real,dim> copy3(n_now_3,order3);
  thrust::gather(order3.begin(),order3.end(),
                 n_now_3.begin(), n_now_4.begin());

  quakins::ReorderCopy<Nums,Real,dim/2> dens_copy({nx1tot,nx2},{1,0});

  FreeStream<Nums,Real,dim,2,0> fsSolverX1(p,p->dt*.5);
  FreeStream<Nums,Real,dim,3,1> fsSolverX2(p,p->dt*.5);

  quakins::BoundaryCondition<Nums,PeriodicBoundary>
    boundX1(nx1,nx1bd);

  quakins::BoundaryCondition<Nums,PeriodicBoundaryPara>
    boundX2(nx2,nx2bd);

  quakins::PoissonSolver<Nums,Real,2, FFTandInvHost<Nums,Real,2>> 
    poissonSolver({nx1,nx2},{x1min,x2min, x1max,x2max});

  quakins::Integrator<Real> 
    integral1(nv1,p->n_1d_per_dev/nv1,v1min,v1max);
  quakins::Integrator<Real> 
    integral2(nv2,p->n_1d_per_dev/nv1/nv2,v2min,v2max);

//--------------------------------------------------------------------
  quakins::PhaseSpaceInitialization
          <Nums,Real,dim>  phaseSpaceInit(p);
  phaseSpaceInit(thrust::device, 
                 f_e.begin(), p->n_1d_per_dev, mpi_rank);
  integral1(f_e.begin(),intg_buff.begin());
  integral2(intg_buff.begin(),dens_e.begin());
    

  std::ofstream dout("dens_e@"+std::to_string(mpi_rank)+".qout",std::ios::out);
  std::ofstream pout("potential@"+std::to_string(mpi_rank)+".qout",std::ios::out);
  //dout << dens_e << std::endl;

  Nums id = mpi_rank;
  Nums l_rank = id==0? mpi_size-1 : id-1;
  Nums r_rank = id==mpi_size-1? 0 : id+1;
  
  char flag;
  if (id==0) flag='l'; 
  else if (id==mpi_size-1) flag='r';
  else flag='m';
  
  Timer the_watch(mpi_rank,"This run");
  Timer push_watch(mpi_rank,"push");
  Timer nccl_watch(mpi_rank,"nccl communination");

  the_watch.tick("Main Loop start...");
  for (Nums step=0; step<p->time_step_total; step++) {
    thrust::copy(f_e.end()-2*comm_size,f_e.end()-comm_size, 
                 r_send_buff.begin());
    thrust::copy(f_e.begin()+comm_size,f_e.begin()+2*comm_size, 
                 l_send_buff.begin());
    nccl_watch.tick("NCCL communicating..."); //----------------------------------
    ncclGroupStart();// <--
    ncclSend(thrust::raw_pointer_cast(l_send_buff.data()),
             comm_size, ncclFloat, l_rank, comm, s); 
    ncclRecv(thrust::raw_pointer_cast(r_recv_buff.data()),
             comm_size, ncclFloat, r_rank, comm, s); 
    ncclGroupEnd();

    ncclGroupStart();// -->
    ncclSend(thrust::raw_pointer_cast(r_send_buff.data()),
               comm_size, ncclFloat, r_rank, comm, s); 
    ncclRecv(thrust::raw_pointer_cast(l_recv_buff.data()),
               comm_size, ncclFloat, l_rank, comm, s); 
    ncclGroupEnd();
    hipStreamSynchronize(s);
    nccl_watch.tock(); //=========================================================

    push_watch.tick("--> step[" +std::to_string(step)+ "] pushing..."); //--------
    
    thrust::copy(l_recv_buff.begin(),l_recv_buff.end(),
                 f_e.begin());
    thrust::copy(r_recv_buff.begin(),r_recv_buff.end(),
                 f_e.end()-comm_size);
      
    copy1(f_e.begin(),f_e.end(),f_e_buff.begin());
    boundX1(f_e_buff.begin(),f_e_buff.end(),flag);
    fsSolverX1(f_e_buff.begin(),
               f_e_buff.end(),
               p->n_1d_per_dev/p->n_tot_local[0],id);
    copy2(f_e_buff.begin(),f_e_buff.end(),f_e.begin());
    //boundX2(f_e.begin(),f_e.end(),flag);
    fsSolverX2(f_e.begin(),
               f_e.end(),
               p->n_1d_per_dev/p->n_tot_local[1],id);
    copy3(f_e.begin(),f_e.end(),f_e_buff.begin());
    thrust::copy(f_e_buff.begin(),f_e_buff.end(),f_e.begin());
    hipStreamSynchronize(s);

    integral1(f_e.begin(),intg_buff.begin());
    integral2(intg_buff.begin(),dens_e.begin());
    
    push_watch.tock(); //==========================================================

    ncclGroupStart();
    if (mpi_rank==0) {
      for (int r=0; r<mpi_size; r++)
        ncclRecv(thrust::raw_pointer_cast(dens_e_tot.data())
                 +nx1tot*nx2bd+r*nxtot, dens_size,ncclFloat,r,comm,s);
    }
    ncclSend(thrust::raw_pointer_cast(dens_e.data())
             +nx1tot*nx2bd, dens_size,ncclFloat,0,comm,s);
    ncclGroupEnd();
    dens_copy(dens_e_tot.begin(),dens_e_tot.end(),dens_e_tot_buff.begin());
  
    if (mpi_rank==0) {
      thrust::copy(dens_e_tot_buff.begin()+nx2/p->n_dev*nx1bd,
                   dens_e_tot_buff.end()-nx2/p->n_dev*nx1bd,
                   _dens_e_tot.begin());
      poissonSolver(_dens_e_tot.begin(),_dens_e_tot.end(),_pote_tot.begin());
    }
    if (mpi_rank==0 && step%(p->dens_print_intv)==0) {
      dout << dens_e_tot << std::endl;
      pout << _pote_tot << std::endl;
    }
  }
  
  the_watch.tock();
  dout.close();

}


