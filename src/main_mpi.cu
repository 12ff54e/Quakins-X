/* -------------------------------
 *     Main of the Quakins-X Code
 * ------------------------------- */

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/gather.h>
#include <cstdio>
#include <mpi.h>
#include <nccl.h>
#include <hipfft/hipfftXt.h>

#include "include/initialization.hpp"
#include "include/PhaseSpaceInitialization.hpp"
#include "include/Timer.hpp"
#include "include/ReorderCopy.hpp"
#include "include/FreeStream.hpp"
#include "include/BoundaryCondition.hpp"
#include "include/Integrator.hpp"
#include "include/PoissonSolver.hpp"
#include "include/util.hpp"
#include "include/details/free_stream_algorithm.hpp"


using Nums = std::size_t;
using Real = float;


constexpr Nums dim = 4;

#define MCW MPI_COMM_WORLD

int main(int argc, char* argv[]) {

  int mpi_rank, mpi_size, local_rank=0;
  MPI_Init(&argc, &argv);
  MPI_Comm_rank(MCW, &mpi_rank);
  MPI_Comm_size(MCW, &mpi_size);

  uint64_t host_hashs[mpi_size];
  char hostname[1024];
  getHostName(hostname, 1024);
  host_hashs[mpi_rank] = getHostHash(hostname);
  MPI_Allgather(MPI_IN_PLACE,0,MPI_DATATYPE_NULL,host_hashs,
                sizeof(uint64_t), MPI_BYTE, MCW);
  for (int p=0; p<mpi_size; p++) {
    if (p==mpi_rank) break;
    if (host_hashs[p]==host_hashs[mpi_rank]) local_rank++;
  }

  ncclUniqueId nccl_id;
  ncclComm_t comm;
  hipStream_t s;

  if (mpi_rank==0) ncclGetUniqueId(&nccl_id);
  MPI_Bcast((void*)&nccl_id,sizeof(nccl_id),MPI_BYTE,0,MCW);
  
  hipSetDevice(local_rank);
  hipStreamCreate(&s);
  
  ncclCommInitRank(&comm,mpi_size,nccl_id,mpi_rank);

  Parameters<Nums,Real,dim> *p = 
             new Parameters<Nums,Real,dim>;

  try { quakins::init(p, mpi_rank); }
  catch (std::invalid_argument& e) 
  {
    std::cerr << e.what() << std::endl;
#ifndef ANYHOW
    return -1;
#endif
  }
  
  Nums nx1=p->n[2], nx2=p->n[3], nx2loc=nx2/p->n_dev;
  Nums nx1bd=p->n_ghost[2], nx2bd=p->n_ghost[3];
  Nums nx1all=p->n_all[2], nx2allloc=p->n_all_local[3];
  Nums nv1=p->n[0], nv2=p->n[1];
  Nums nxall = nx1all*nx2allloc;
  Nums comm_size = p->n_ghost[3]*nx1all*nv1*nv2;
  Nums dens_size = nx1all*nx2/p->n_dev;
  
  Real v1min=p->low_bound[0], v1max=p->up_bound[0];
  Real v2min=p->low_bound[1], v2max=p->up_bound[1];
  Real x1min=p->low_bound[2], x1max=p->up_bound[2];
  Real x2min=p->low_bound[3], x2max=p->up_bound[3];

  thrust::device_vector<Real> 
    l_send_buff(comm_size), l_recv_buff(comm_size), 
    r_send_buff(comm_size), r_recv_buff(comm_size); 
  thrust::device_vector<Real> 
    f_e(p->n_1d_per_dev), f_e_buff(p->n_1d_per_dev);
  thrust::device_vector<Real> 
    intg_buff(nxall*nv2), dens_e(nxall);
  thrust::device_vector<Real> 
    dens_e_all(nx1all*nx2), dens_e_all_buff(nx1all*nx2), pote_all(nx1*nx2);
  thrust::host_vector<Real> _dens_e_all(nx1*nx2), _pote_all(nx1*nx2);

  std::array<Nums,4> order1 = {3,2,0,1},
                     order2 = {1,0,3,2},
                     order3 = {3,2,0,1};

  std::array<Nums,4> n_now_1 = p->n_all_local;
  std::array<Nums,4> n_now_2, n_now_3, n_now_4;
  quakins::ReorderCopy<Nums,Real,dim> copy1(n_now_1,order1);
  thrust::gather(order1.begin(),order1.end(),
                 n_now_1.begin(), n_now_2.begin());
  quakins::ReorderCopy<Nums,Real,dim> copy2(n_now_2,order2);
  thrust::gather(order2.begin(),order2.end(),
                 n_now_2.begin(), n_now_3.begin());
  quakins::ReorderCopy<Nums,Real,dim> copy3(n_now_3,order3);
  thrust::gather(order3.begin(),order3.end(),
                 n_now_3.begin(), n_now_4.begin());

  quakins::ReorderCopy<Nums,Real,dim/2> dens_copy({nx1all,nx2},{1,0});

  quakins::FreeStream<Nums,Real,dim,2,0,
    quakins::details::FluxBalance> fsSolverX1(p,p->dt*.5);
  quakins::FreeStream<Nums,Real,dim,3,1,
    quakins::details::FluxBalance> fsSolverX2(p,p->dt*.5);

  quakins::BoundaryCondition<Nums,ReflectingBoundary>
    boundX1(nx1,nx1bd,nv1,nx1all*nx2allloc*nv2);


  quakins::PoissonSolver<Nums,Real,2, FFTandInvHost> 
    poissonSolver({nx1,nx2},{x1min,x2min, x1max,x2max});

  quakins::Integrator<Real> 
    integral1(nv1,p->n_1d_per_dev/nv1,v1min,v1max);
  quakins::Integrator<Real> 
    integral2(nv2,p->n_1d_per_dev/nv1/nv2,v2min,v2max);

//--------------------------------------------------------------------
  quakins::PhaseSpaceInitialization
          <Nums,Real,dim>  phaseSpaceInit(p);
  phaseSpaceInit(thrust::device, 
                 f_e.begin(), p->n_1d_per_dev, mpi_rank);
  integral1(f_e.begin(),intg_buff.begin());
  integral2(intg_buff.begin(),dens_e.begin());
    

  std::ofstream dout("dens_e@"+std::to_string(mpi_rank)+".qout",std::ios::out);
  std::ofstream pout("potential@"+std::to_string(mpi_rank)+".qout",std::ios::out);
  //dout << dens_e << std::endl;

  ncclGroupStart();
  if (mpi_rank==0) {
    for (int r=0; r<mpi_size; r++)
      ncclRecv(thrust::raw_pointer_cast(dens_e_all.data())
               +r*dens_size, dens_size,ncclFloat,r,comm,s);
  }
  ncclSend(thrust::raw_pointer_cast(dens_e.data())
           +nx1all*nx2bd, dens_size,ncclFloat,0,comm,s);
  ncclGroupEnd();
 
  if (mpi_rank==0) {
    dens_copy(dens_e_all.begin(),dens_e_all.end(),dens_e_all_buff.begin());
    thrust::copy(dens_e_all_buff.begin()+nx2*nx1bd,
                 dens_e_all_buff.end()-nx2*nx1bd,
                 _dens_e_all.begin());
    poissonSolver(_dens_e_all.begin(),_dens_e_all.end(),_pote_all.begin());
    dout << _dens_e_all << std::endl;
    pout << _pote_all << std::endl;
 
  }


  Nums id = mpi_rank;
  Nums l_rank = id==0? mpi_size-1 : id-1;
  Nums r_rank = id==mpi_size-1? 0 : id+1;
  
  char flag;
  if (id==0) flag='l'; 
  else if (id==mpi_size-1) flag='r';
  else flag='m';
  
  Timer the_watch(mpi_rank,"This run");
  Timer push_watch(mpi_rank,"push");
  Timer nccl_watch(mpi_rank,"nccl communination");
  Timer poi_watch(mpi_rank,"solver Poisson equation");
  /*
  std::ofstream fout("ftest@" +std::to_string(mpi_rank)+ ".qout",std::ios::out);
  fout << f_e << std::endl;
*/
  the_watch.tick("Main Loop start...");
  for (Nums step=0; step<p->time_step_total; step++) {

    thrust::copy(f_e.end()-2*comm_size,f_e.end()-comm_size, 
                 r_send_buff.begin());
    thrust::copy(f_e.begin()+comm_size,f_e.begin()+2*comm_size, 
                 l_send_buff.begin());
    nccl_watch.tick("NCCL communicating..."); //----------------------------------
    ncclGroupStart();// <--
    ncclSend(thrust::raw_pointer_cast(l_send_buff.data()),
             comm_size, ncclFloat, l_rank, comm, s); 
    ncclRecv(thrust::raw_pointer_cast(r_recv_buff.data()),
             comm_size, ncclFloat, r_rank, comm, s); 
    ncclGroupEnd();

    ncclGroupStart();// -->
    ncclSend(thrust::raw_pointer_cast(r_send_buff.data()),
             comm_size, ncclFloat, r_rank, comm, s); 
    ncclRecv(thrust::raw_pointer_cast(l_recv_buff.data()),
             comm_size, ncclFloat, l_rank, comm, s); 
    ncclGroupEnd();

    hipStreamSynchronize(s);

    
    thrust::copy(l_recv_buff.begin(),l_recv_buff.end(),
                 f_e.begin());
    thrust::copy(r_recv_buff.begin(),r_recv_buff.end(),
                 f_e.end()-comm_size);
    nccl_watch.tock(); //=========================================================
      
    push_watch.tick("--> step[" +std::to_string(step)+ "] pushing..."); //--------
    copy1(f_e.begin(), f_e.end(),f_e_buff.begin()); // n_now = {nx2l,nx1,nv1,nv2}
    fsSolverX2(f_e_buff.begin(),
               f_e_buff.end(),
               p->n_1d_per_dev/p->n_all_local[1],id);
    copy2(f_e_buff.begin(),f_e_buff.end(),f_e.begin()); // n_now = {nx1,nx2l,nv2,nv1}

    boundX1(f_e.begin(),f_e.end(),flag);
    fsSolverX1(f_e.begin(),
               f_e.end(),
               p->n_1d_per_dev/p->n_all_local[0],id);
    copy3(f_e.begin(),f_e.end(),f_e_buff.begin()); // n_now = {nv1,nv2,nx1,nx2l}

    thrust::copy(f_e_buff.begin(),f_e_buff.end(),f_e.begin());

    hipStreamSynchronize(s);

    integral1(f_e.begin(),intg_buff.begin());
    integral2(intg_buff.begin(),dens_e.begin());
    
    push_watch.tock(); //==========================================================

    poi_watch.tick("solving poisson...");

    ncclGroupStart();
    if (mpi_rank==0) {
      for (int r=0; r<mpi_size; r++)
        ncclRecv(thrust::raw_pointer_cast(dens_e_all.data())
                 +r*dens_size, dens_size,ncclFloat,r,comm,s);
    }
    ncclSend(thrust::raw_pointer_cast(dens_e.data())
             +nx1all*nx2bd, dens_size,ncclFloat,0,comm,s);
    ncclGroupEnd();
  
    if (mpi_rank==0) {
      dens_copy(dens_e_all.begin(),dens_e_all.end(),dens_e_all_buff.begin());
      thrust::copy(dens_e_all_buff.begin()+nx2*nx1bd,
                   dens_e_all_buff.end()-nx2*nx1bd,
                   _dens_e_all.begin());
      poissonSolver(_dens_e_all.begin(),_dens_e_all.end(),_pote_all.begin());
    }
    poi_watch.tock();

    if (mpi_rank==0 && step%(p->dens_print_intv)==0) {
      dout << _dens_e_all << std::endl;
      pout << _pote_all << std::endl;
    }
  }
  

  the_watch.tock();
  dout.close();


}


